
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<vector>
using namespace std;
#define M 175000000
#define N 2*M
#define S 8

__global__ void square(char I[M], unsigned int O[N]){
	int x0 = blockIdx.x;
	int x1 = blockIdx.y;
	int x2 = blockIdx.z;
	int x3 = threadIdx.x;
	int size=0;
	int it=0;
	int pow=1;
	while(I[it]!='+'){
		size+=(I[it]-48)*pow;
		it++;
		pow=pow*10;
	}
	it++;
	int idx=x0+x1*S+x2*S*S+x3*S*S*S;
	
	if(idx<2*size-1){	
		int i,j;
		if(idx<size){
			i=size-1-idx;
			j=size-1;
		}else{
			i=0;
			j=size-1-(idx-(size-1));
		}
		while(i!=size && j!=-1 && i<=j){
			if(i<j)
				O[idx]+=2*(I[it+i]-48)*(I[it+j]-48);
			else if(i==j){
				O[idx]+=(I[it+i]-48)*(I[it+j]-48);
			}
			i++;
			j--;
		}
	}

	O[N-1]=1;
}

int main(){
	fstream fin,foutr;
	string file="s.txt";
	fin.open(file.c_str());
	foutr.open("r.txt");

	char *hostI=new char[M];
	unsigned int *hostO=new unsigned int[N];

	int size=0;
	int it=0;
	int pow=1;
	char s;
	fin>>s;
	while(s!='+'){
		size+=(s-48)*pow;
		it++;
		pow=pow*10;
		fin>>s;
	}
	it++;

	fin.close();
	fin.open(file.c_str());

	cout<<"input"<<endl;
	for(int i=0;i<size+it;i++){
		fin>>s;
		hostI[i]=s;
		cout<<hostI[i]<<endl;
	}


	char *I;

	unsigned int *O;

	hipMalloc((void**)&I, sizeof(char) * M);

	hipMalloc((void**)&O, sizeof(unsigned int) * N);

	hipMemcpy(I,hostI,sizeof(char) * M,hipMemcpyHostToDevice);

	hipMemcpy(O,hostO,sizeof(unsigned int) * N,hipMemcpyHostToDevice);

	dim3 blocks(S,S,S);
	dim3 threads(S,1,1);

	square<<<blocks,threads>>>((char(*))I, (unsigned int(*))O);

	hipMemcpy(hostO,O,sizeof(unsigned int) * N,hipMemcpyDeviceToHost);

	cout<<"output"<<endl;
	for (int i=0;i<2*size-1;i++){
		cout<<hostO[i]<<endl;
	}

	cout<<hostO[N-1]<<endl;

	return 0;

}