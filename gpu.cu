
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<vector>
using namespace std;
#define M 9000000
#define N 2*M
#define B 1024
#define T 8

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int size=I[0];
	int idx_start=(y+x*T);
	int idx_end=(1+y+x*T);
	int C=B*T;
	if(2*size>C){
		int jump=2*size/(C/2)-1;
		if(idx_start<(C/4)){
			idx_start*=jump;
			idx_end*=jump;
		}else if(idx_start>(C/2)+(C/4)-1){
			idx_start-=((C/2)+(C/4));
			idx_start*=jump;
			idx_start+=((C/4)*jump+(C/2));
			idx_end-=((C/2)+(C/4));
			idx_end*=jump;
			idx_end+=((C/4)*jump+(C/2));
		}else{
			idx_start-=(C/4);
			idx_start+=(C/4)*jump;
			idx_end-=(C/4);
			idx_end+=(C/4)*jump;
		}
	}
	
	for(int idx=idx_start+1;idx<idx_end+1;idx++){
		if(idx<2*size){	
			int i,j;
			if(idx<size+1){
				i=idx;
				j=1;
			}else{
				i=size;
				j=idx%size;
			}
			while(i!=0 && j!=size+1 && i>=j){
				if(i>j){
					O[idx]+=2*(I[i])*(I[j]);
				}
				else if(i==j){
					O[idx]+=(I[i])*(I[j]);
				}
				i--;
				j++;
			}
		}
	}
	if(idx_end==size*2)
		// printf("index_end : %d\n", idx_end);
	O[N-1]=1;
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	int size=1;
	int base=1024*1024;

	for(int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else{
			hostI[i]=4;
		}
	}


	unsigned long long int *I;

	unsigned long long int *O;
	
	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	for(int i=0;i<5;i++){
		// cout<<i<<endl;
		hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);

		hipMemcpy(O,hostO,sizeof(unsigned long long int) * (2*size),hipMemcpyHostToDevice);
		
		dim3 blocks(B,1,1);
		dim3 threads(T,1,1);

		square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);
		
		hipMemcpy(hostO,O,sizeof(unsigned long long int) * N,hipMemcpyDeviceToHost);
		unsigned long long int c=0;
		int pos=1;
		int flag=0;
		while (c!=0 || pos<2*size){
			if(pos>=2*size)
				hostO[pos]=0;
			hostO[pos]=hostO[pos]+c;
			c=hostO[pos]/base;
			hostO[pos]=hostO[pos]%base;
			
			if(pos==1){
				// cout<<"number : ";
				if(hostO[pos]<2){
					hostO[pos]=base-2+hostO[pos];
					flag=1;
				}else{
					hostO[pos]-=2;
				}
			}else if(flag==1 && hostO[pos]==0){
				hostO[pos]=base-1;
			}else if(flag==1){
				hostO[pos]-=1;
				flag=0;
			}
			// cout<<hostO[pos]<<" ";
			hostI[pos]=hostO[pos];
			hostO[pos]=0;
			if(c!=0 || hostI[pos]!=0)
				pos++;
		}
		hostO[0]=pos-1;
		hostI[0]=hostO[0];
		size=hostI[0];
		hostO[0]=0;
		cout<<endl<<"size : "<<hostI[0]<<endl;
		cout<<"working : "<<hostO[N-1]<<endl<<endl;
	}
	return 0;
}