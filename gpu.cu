
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<vector>
using namespace std;
#define M 9000000
#define N 2*M-1
#define B 1024
#define T 8

__global__ void square(char I[M], unsigned int O[N]){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int size=0;
	int it=0;
	int pow=1;
	while(I[it]!='+'){
		size+=(I[it]-48)*pow;
		it++;
		pow=pow*10;
	}
	it++;
	int idx_start=(y+x*T);
	int idx_end=(1+y+x*T);
	int C=B*T;
	if(2*size>C){
		int jump=2*size/(C/2)-1;
		if(idx_start<(C/4)){
			idx_start*=jump;
			idx_end*=jump;
		}else if(idx_start>(C/2)+(C/4)-1){
			idx_start-=((C/2)+(C/4));
			idx_start*=jump;
			idx_start+=((C/4)*jump+(C/2));
			idx_end-=((C/2)+(C/4));
			idx_end*=jump;
			idx_end+=((C/4)*jump+(C/2));
		}else{
			idx_start-=(C/4);
			idx_start+=(C/4)*jump;
			idx_end-=(C/4);
			idx_end+=(C/4)*jump;
		}
	}
	
	for(int idx=idx_start;idx<idx_end;idx++){
		if(idx<2*size){	
			int i,j;
			if(idx<size){
				i=size-1-idx;
				j=size-1;
			}else{
				i=0;
				j=size-1-(idx-(size-1));
			}
			while(i!=size && j!=-1 && i<=j){
				if(i<j)
					O[idx]+=2*(I[it+i]-48)*(I[it+j]-48);
				else if(i==j){
					O[idx]+=(I[it+i]-48)*(I[it+j]-48);
				}
				i++;
				j--;
			}
		}
	}
	if(idx_end==size*2)
		printf("%d\n", idx_end);
	O[N-1]=1;
}
/*
void base_case(fstream &foutr,int v,int &h,const unsigned int b){
	if(v!=0){
		base_case(foutr,v/b,h,b);
		foutr<<v%b;
		h++;
	}
	return;
}

void add(fstream &foutr,int* v,int c,int i,int h,int size,const unsigned int b){
	if(i==2*(size-1)){
		base_case(foutr,v,h,b);
		return;
	}else{
		v=v+c;
		c=v/b;
		char out=48+(v%b);
		foutr<<out;
		h++;
		add(foutr,v,c,i,h,size,b);
		return;
	}
}
*/
int main(){
	fstream fin,foutr;
	string file="s.txt";
	fin.open(file.c_str());
	foutr.open("r.txt");

	char *hostI=new char[M];
	unsigned int *hostO=new unsigned int[N];

	int size=0;
	int it=0;
	int pow=1;
	char s;
	fin>>s;
	while(s!='+'){
		size+=(s-48)*pow;
		it++;
		pow=pow*10;
		fin>>s;
	}
	it++;

	fin.close();
	fin.open(file.c_str());

	//cout<<"input"<<endl;
	for(int i=0;i<size+it;i++){
		fin>>s;
		hostI[i]=s;
		//cout<<hostI[i]<<endl;
	}


	char *I;

	unsigned int *O;
	cout<<"before alloc"<<endl;
	hipMalloc((void**)&I, sizeof(char) * M);

	hipMalloc((void**)&O, sizeof(unsigned int) * N);
	// cout<<"after alloc"<<endl;
	// cout<<"before copy"<<endl;
	hipMemcpy(I,hostI,sizeof(char) * (size+it),hipMemcpyHostToDevice);

	hipMemcpy(O,hostO,sizeof(unsigned int) * 2*size-1,hipMemcpyHostToDevice);
	cout<<"after copy"<<endl;
	dim3 blocks(B,1,1);
	dim3 threads(T,1,1);

	square<<<blocks,threads>>>((char(*))I, (unsigned int(*))O);
	
	hipMemcpy(hostO,O,sizeof(unsigned int) * N,hipMemcpyDeviceToHost);
	
	// cout<<"output"<<endl;
	for (int i=0;i<2*size-1;i++){
		cout<<hostO[i]<<" ";
	}
	
	// add(&hostO,foutr,);

	cout<<hostO[N-1]<<endl;

	return 0;

}