
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<vector>
using namespace std;
#define M 9000000
#define N 2*M
#define B 1024
#define T 8

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int size=I[0];
	int idx_start=(y+x*T);
	int idx_end=(1+y+x*T);
	int C=B*T;
	if(2*size>C){
		int jump=2*size/(C/2)-1;
		if(idx_start<(C/4)){
			idx_start*=jump;
			idx_end*=jump;
		}else if(idx_start>(C/2)+(C/4)-1){
			idx_start-=((C/2)+(C/4));
			idx_start*=jump;
			idx_start+=((C/4)*jump+(C/2));
			idx_end-=((C/2)+(C/4));
			idx_end*=jump;
			idx_end+=((C/4)*jump+(C/2));
		}else{
			idx_start-=(C/4);
			idx_start+=(C/4)*jump;
			idx_end-=(C/4);
			idx_end+=(C/4)*jump;
		}
	}
	
	for(int idx=idx_start;idx<idx_end;idx++){
		if(idx<2*size){	
			int i,j;
			if(idx<size){
				i=size-1-idx;
				j=size-1;
			}else{
				i=0;
				j=size-1-(idx-(size-1));
			}
			while(i!=size && j!=-1 && i<=j){
				if(i<j)
					O[idx+1]+=2*(I[1+i])*(I[1+j]);
				else if(i==j){
					O[idx+1]+=(I[1+i])*(I[1+j]);
				}
				i++;
				j--;
			}
		}
	}
	if(idx_end==size*2)
		printf("%d\n", idx_end);
	O[N-1]=1;
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	int size=4096;

	for(int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else{
			hostI[i]=1;
		}
	}


	unsigned long long int *I;

	unsigned long long int *O;
	cout<<"before alloc"<<endl;
	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	// cout<<"after alloc"<<endl;
	// cout<<"before copy"<<endl;
	hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);

	hipMemcpy(O,hostO,sizeof(unsigned long long int) * (2*size),hipMemcpyHostToDevice);
	cout<<"after copy"<<endl;
	dim3 blocks(B,1,1);
	dim3 threads(T,1,1);

	square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);
	
	hipMemcpy(hostO,O,sizeof(unsigned long long int) * N,hipMemcpyDeviceToHost);
	
	// cout<<"output"<<endl;
	for (int i=0;i<2*size;i++){
		// cout<<hostO[i]<<" ";
	}
	
	cout<<hostO[N-1]<<endl;

	return 0;

}