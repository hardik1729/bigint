
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
using namespace std;
#define M 17000000
#define N 2*M
const long int B = 2097152;
const long int T = 1024;

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int size=I[0];
	int idx_start=(y+x*T);
	int idx_end=(1+y+x*T);
	long int C=B*T;

	if(2*size-1>C){
		int n=int((sqrtf(1+2*(2*size-1-C))-1)/2)+1;
		int rem=2*size-1-C-2*n*(n-1);
		int rem_l=rem/2;
		int rem_r=rem-rem_l;
		if(idx_start<n || idx_end>C-n){
			if(idx_start<C/2){
				int idx=idx_start;
				idx_start=(n*n-(n-idx)*(n-idx));
				idx_end=idx_start+(1+(n-idx-1)*2);
			}else{
				idx_start-=(C-n);
				int idx=idx_start;
				idx_start=(n*n+C-2*n+idx*idx);
				idx_end=idx_start+(1+idx*2);
			}
		}else{
			idx_start-=n;
			idx_start+=(n*n);
			idx_end-=n;
			idx_end+=(n*n);
		}	
		if(idx_start==0){
			idx_end+=rem_l;
		}else if(idx_end==2*size-1-(rem_l+rem_r)){
			idx_start+=rem_l;
			idx_end+=(rem_r+rem_l);
		}else{
			idx_start+=rem_l;
			idx_end+=rem_l;
		}
	}

	for(int idx=idx_start+1;idx<idx_end+1;idx++){
		if(idx<2*size){	
			O[idx]=0;
			int i,j;
			if(idx<size+1){
				i=idx;
				j=1;
			}else{
				i=size;
				j=(idx%size)+1;
			}
			while(i!=0 && j!=size+1 && i>=j){
				if(i>j){
					O[idx]+=2*(I[i])*(I[j]);
				}
				else if(i==j){
					O[idx]+=(I[i])*(I[j]);
				}
				i--;
				j++;
			}
		}
	}
	if(idx_end==size*2-1)
		printf("index_end : %d\n", idx_end);
	O[N-1]=1;
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	int size=140;
	int base=1024*1024;

	for(int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else if(i==size){
			hostI[i]=1;
		}else{
			hostI[i]=0;
		}
	}


	unsigned long long int *I;

	unsigned long long int *O;
	
	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	while(size<=140){
		hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);

		hipMemcpy(O,hostO,sizeof(unsigned long long int) * (2*size),hipMemcpyHostToDevice);
		
		dim3 blocks(B,1,1);
		dim3 threads(T,1,1);

		square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);
		
		hipMemcpy(hostO,O,sizeof(unsigned long long int) * N,hipMemcpyDeviceToHost);
		unsigned long long int c=0;
		int pos=1;
		int flag=0;
		while (c!=0 || pos<2*size){
			if(pos>=2*size)
				hostO[pos]=0;
			hostO[pos]=hostO[pos]+c;
			c=hostO[pos]/base;
			hostO[pos]=hostO[pos]%base;
			/*if(pos==1){
				if(hostO[pos]<2){
					hostO[pos]=base-2+hostO[pos];
					flag=1;
				}else{
					hostO[pos]-=2;
				}
			}else if(flag==1 && hostO[pos]==0){
				hostO[pos]=base-1;
			}else if(flag==1){
				hostO[pos]-=1;
				flag=0;
			}*/
			hostI[pos]=hostO[pos];
			// cout<<hostO[pos]<<",";
			hostO[pos]=0;
			if((pos>=2*size && (c!=0 || hostI[pos]!=0)) || pos<2*size)
				pos++;
		}
		if(hostI[pos-1]==0){
			pos--;
		}
		hostO[0]=pos-1;
		hostI[0]=hostO[0];
		size=hostI[0];
		hostO[0]=0;
		cout<<"size : "<<hostI[0]<<","<<hostI[hostI[0]]<<endl;
		cout<<"working : "<<hostO[N-1]<<endl<<endl;
	}
	return 0;
}
