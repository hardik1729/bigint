
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
using namespace std;
const long int M = 17000000;
const long int D = 2;
const long int N = D*(2*M-1);
const long int B = 2097152;
const long int T = 1024;

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int size=I[0];
        int idx=(y+x*T);
        if(idx<D*(2*size-1)){
                O[idx+1]=0;
                int i,j,op,idx_one,idx_two;
                if(idx/D<size){
                        i=idx/D+1;
                        j=1;
                        op=(i+1)/2;                                                              }else{
                        i=size;
                        j=((idx/D)%size)+2;
                        op=(size-j)/2+1;                                                         }                                                                                if(op>=D){
                        i-=(op/D)*(idx%D);
                        j+=(op/D)*(idx%D);                                                               if(idx%D==D-1){
                                idx_one=i+(op/D)*(idx%D)-op;                                                     idx_two=j-(op/D)*(idx%D)+op;
                        }else{
                                idx_one=i-op/D;
                                idx_two=j+op/D;
                        }
                }else{
                        if(idx%D!=D-1){                                                                          idx_one=i;
                                idx_two=j;
                        }else{                                                                                   idx_one=i-op;                                                                    idx_two=j+op;                                                            }
                }
                while(i!=idx_one && j!=idx_two){
                        if(i>j){
                                O[idx+1]+=2*(I[i])*(I[j]);
                        }
                        else if(i==j){
                                O[idx+1]+=(I[i])*(I[j]);
                        }
                        i--;
                        j++;
                }
        }
	if(idx+1==(2*size-1)*D)
		printf("index_end : %d\n", idx+1);
	O[N-1]=1;
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	int size=4;
	int base=1024*1024;

	for(int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else if(i==size){
			hostI[i]=1;
		}else{
			hostI[i]=0;
		}
	}

	unsigned long long int *I;

	unsigned long long int *O;

	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	while(size<=4){
		hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);

		hipMemcpy(O,hostO,sizeof(unsigned long long int) * (2*size-1)*D,hipMemcpyHostToDevice);
		
		dim3 blocks(B,1,1);
		dim3 threads(T,1,1);

		square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);
		hipMemcpy(hostO,O,sizeof(unsigned long long int) * N,hipMemcpyDeviceToHost);
		unsigned long long int c=0;
		int pos=1;
		int flag=0;
		while (c!=0 || pos<2*size){
			hostI[pos]=0;
			if(pos>2*size-1){
				for(int pos_sub=D*(pos-1)+1;pos_sub<D*pos+1;pos_sub++)
					hostO[pos_sub]=0;
			}
			for(int pos_sub=D*(pos-1)+1;pos_sub<D*pos+1;pos_sub++)
				hostI[pos]+=hostO[pos_sub];
			
			hostI[pos]=hostI[pos]+c;
			c=hostI[pos]/base;
			hostI[pos]=hostI[pos]%base;
			/*if(pos==1){
				if(hostI[pos]<2){
					hostI[pos]=base-2+hostI[pos];
					flag=1;
				}else{
					hostI[pos]-=2;
				}
			}else if(flag==1 && hostI[pos]==0){
				hostI[pos]=base-1;
			}else if(flag==1){
				hostI[pos]-=1;
				flag=0;
			}
			*/
			//cout<<hostI[pos]<<","<<c<<";";
			if((pos>=2*size && (c!=0 || hostI[pos]!=0)) || pos<2*size)
				pos++;
		}
		if(hostI[pos-1]==0){
			pos--;
		}
		hostI[0]=pos-1;
		size=hostI[0];
		cout<<"size : "<<hostI[0]<<","<<hostI[hostI[0]]<<endl;
		cout<<"working : "<<hostO[N-1]<<endl<<endl;
	}
	return 0;
}
