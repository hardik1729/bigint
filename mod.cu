
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
using namespace std;
#define M (100+1)
#define N 2*(M-1)
unsigned long long int B = 65535;
unsigned long long int T = 1024;

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	unsigned long long int x = blockIdx.x;
	unsigned long long int y = threadIdx.x;
	unsigned long long int size=I[0];	
	unsigned long long int T=1024;
	unsigned long long int B=1+(2*size-1)/T;
	unsigned long long int idx_start=(y+x*T);
	unsigned long long int idx_end=(1+y+x*T);
	long int C=B*T;

	for(unsigned long long int idx=idx_start+1;idx<idx_end+1;idx++){
		if(idx<2*size){
			O[idx]=0;
			unsigned long long int i,j;
			if(idx<size+1){
				i=idx;
				j=1;
			}else{
				i=size;
				j=(idx%size)+1;
			}
			while(i!=0 && j!=size+1 && i>=j){
				if(i>j){
					O[idx]+=2*(I[i])*(I[j]);
				}
				else if(i==j){
					O[idx]+=(I[i])*(I[j]);
				}
				i--;
				j++;
			}
		}
	}
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	unsigned long long int size=1;
	unsigned long long int s=20;
	unsigned long long int base=pow(2,s);
	unsigned long long int p;
	cin>>p;
	int total_count=p-2;
	int count=0;

	for(unsigned long long int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else if(i==size){
			hostI[i]=4;
		}else{
			hostI[i]=0;
		}
	}

	unsigned long long int *I;

	unsigned long long int *O;
	
	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	while(count<=total_count){
//SQUARE
		hipError_t err=hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);
		cout<<hipGetErrorString(err)<<endl;

		// err=cudaMemcpy(O,hostO,sizeof(unsigned long long int) * (2*size),cudaMemcpyHostToDevice);
		// cout<<err<<endl;

		T=1024;
		B=1+(2*size-1)/T;

		dim3 blocks(B,1,1);
		dim3 threads(T,1,1);

		square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);	

		err=hipMemcpy(hostO,O,sizeof(unsigned long long int) * 2*size,hipMemcpyDeviceToHost);
		cout<<hipGetErrorString(err)<<endl;
		// for(int i=1;i<2*size;i++)
			// cout<<hostO[i]<<",";
		// cout<<endl;

//NORMALIZED SUM
		int carry=0;
		int idx;
		for(idx=1;carry!=0 || idx<2*size;idx++){
			hostI[idx]=(hostO[idx]+carry)%base;
			carry=(carry+hostO[idx])/base;
			// cout<<hostI[idx]<<",";	
		}
		// cout<<endl;
		hostI[0]=--idx;

//SUBTRACT 2
		int flag=0;
		if(total_count!=count){
			for(int i=1;i<hostI[0]+1;i++){
				if(i==1){
					if(hostI[i]<2){
						flag=1;
					}
					hostI[i]=(hostI[i]+base-2)%base;
				}else if(flag){
					if(hostI[i]<1)
						flag=1;
					else
						flag=0;
					hostI[i]=(hostI[i]+base-1)%base;
				}else{
					break;
				}
				// cout<<hostI[i]<<",";
			}
			// cout<<endl;
			if(hostI[idx]==0)
				hostI[0]--;
		}

//MODULO REDUCTION TO HALF
		cout<<"step : "<<count<<endl;
		int m_count=0;
		while(hostI[0]>p/s+1 && m_count<5){
			flag=0;
			if(2*(p/s+1)-hostI[0]==1)
				flag=1;
			carry=0;
			for(idx=1;carry!=0 || (idx<=hostI[0] && idx<=p/20+1);idx++){
				if(idx>p/s+1){
					hostI[idx]=carry;
					carry=0;
				}else if(idx==p/s+1 && flag){
					hostI[idx]=(hostI[idx]+carry);
					carry=(hostI[idx])/base;
					hostI[idx]%=base;		
				}else{
					hostI[idx]=(hostI[idx]+hostI[idx+p/s+1]*pow(2,s-p%s)+carry);
					carry=(hostI[idx])/base;
					hostI[idx]%=base;
				}
				// cout<<idx<<","<<hostI[idx]<<";";
			}
			// cout<<endl;
			while(--idx>-1 && hostI[idx]==0);
			hostI[0]=idx;
			m_count++;
		}

//MODULO
		// while(hostI[0]==p/20+1 && hostI[hostI[0]]>=pow(2,p%20)){
		// 	cout<<endl<<"comeback after coding it.";
		// }
 		size=hostI[0];
 		for(int i=1;i<size+1;i++)
 			cout<<i-1<<","<<hostI[i]<<";";
 		cout<<endl;
		cout<<"size : "<<hostI[0]<<endl<<endl;
		count++;
	}
	return 0;
}
