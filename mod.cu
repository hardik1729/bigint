
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
using namespace std;
#define M (100000+1)
#define N 2*(M-1)
unsigned long long int B = 65535;
unsigned long long int T = 1024;

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	unsigned long long int x = blockIdx.x;
	unsigned long long int y = threadIdx.x;
	unsigned long long int size=I[0];	
	unsigned long long int T=1024;
	unsigned long long int B=1+(2*size-1)/T;
	unsigned long long int idx_start=(y+x*T);
	unsigned long long int idx_end=(1+y+x*T);
	long int C=B*T;

	for(unsigned long long int idx=idx_start+1;idx<idx_end+1;idx++){
		if(idx<2*size){
			O[idx]=0;
			unsigned long long int i,j;
			if(idx<size+1){
				i=idx;
				j=1;
			}else{
				i=size;
				j=(idx%size)+1;
			}
			while(i!=0 && j!=size+1 && i>=j){
				if(i>j){
					O[idx]+=2*(I[i])*(I[j]);
				}
				else if(i==j){
					O[idx]+=(I[i])*(I[j]);
				}
				i--;
				j++;
			}
		}
	}
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	unsigned long long int size=1;
	unsigned long long int s=3;
	unsigned long long int base=pow(2,s);
	unsigned long long int p;
	cin>>p;
	int total_count=p-2;
	int count=0;

	for(unsigned long long int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else if(i==size){
			hostI[i]=4;
		}else{
			hostI[i]=0;
		}
	}

	unsigned long long int *I;

	unsigned long long int *O;
	
	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	while(count<=total_count){
		cout<<"\r"<<"step : "<<count;
//SQUARE
		hipError_t err=hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);
		if(err)
			cout<<hipGetErrorString(err)<<endl;

		T=1024;
		B=1+(2*size-1)/T;

		dim3 blocks(B,1,1);
		dim3 threads(T,1,1);

		square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);	

		err=hipMemcpy(hostO,O,sizeof(unsigned long long int) * 2*size,hipMemcpyDeviceToHost);
		if(err)
			cout<<hipGetErrorString(err)<<endl;

//NORMALIZED SUM
		unsigned long long int carry=0;
		int idx;
		for(idx=1;carry!=0 || idx<2*size;idx++){
			if(idx<2*size){
				hostI[idx]=(hostO[idx]+carry)%base;
				carry=(carry+hostO[idx])/base;
			}else{
				hostI[idx]=carry%base;
				carry=carry/base;
			}
			// cout<<hostI[idx]<<",";	
		}
		// cout<<endl;
		hostI[0]=--idx;

//SUBTRACT 2
		if(total_count!=count){
			int flag=0;
			for(int i=1;i<hostI[0]+1;i++){
				if(i==1){
					if(hostI[i]<2){
						flag=1;
					}
					hostI[i]=(hostI[i]+base-2)%base;
				}else if(flag){
					if(hostI[i]<1)
						flag=1;
					else
						flag=0;
					hostI[i]=(hostI[i]+base-1)%base;
				}else{
					break;
				}
				// cout<<hostI[i]<<",";
			}
			// cout<<endl;
			if(hostI[idx]==0)
				hostI[0]--;
		}

//MODULO
		while(hostI[0]>p/s+1){
			for(idx=1;carry!=0 || idx<=p/s+1;idx++){
				if(idx>p/s+1){
					hostI[idx]=carry;
					carry=hostI[idx]/base;
					hostI[idx]%=base;
				}else if(idx+p/s+1>hostI[0]){
					hostI[idx]=hostI[idx]+carry;
					carry=hostI[idx]/base;
					hostI[idx]%=base;
				}else{
					hostI[idx]=hostI[idx]+hostI[idx+p/s+1]*pow(2,s-(p%s))+carry;
					carry=hostI[idx]/base;
					hostI[idx]%=base;
				}
				// cout<<idx<<","<<hostI[idx]<<","<<carry<<";";
			}
			// cout<<endl;
			while(--idx && hostI[idx]==0);
			hostI[0]=idx;
		}
//FINAL CHECK
 		size=hostI[0];
 		if(total_count==count){
 			cout<<endl;
 			if(size==p/s+1){
	 			int flag=1;
	 			for(int i=1;i<size+1;i++){
	 				if(i>1 && i<size)
	 					flag=flag && (hostI[i]==base-1);
	 				if(hostI[i]!=base-1)
	 					cout<<i-1<<","<<hostI[i]<<";";
	 			}
	 			cout<<endl;
	 			unsigned long long int l=base-hostI[1];
	 			unsigned long long int h=pow(2,p%s)*l-1;
	 			if(h==hostI[size] && flag)
	 				cout<<"PRIME"<<endl;
	 			else
	 				cout<<"COMPOSITE"<<endl;
	 		}else{
	 			for(int i=1;i<size+1;i++){
					cout<<hostI[i]<<"*"<<"2**"<<s*(i-1)<<"+";
				}
				cout<<endl;
	 			cout<<"TRICKY"<<endl;
	 		}
	 		cout<<"size : "<<hostI[0]<<endl<<endl;
	 	}
 		count++;
	}
	return 0;
}
