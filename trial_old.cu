
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
using namespace std;
#define M (10000+1)
#define N 2*(M-1)
unsigned long long int B = 65535;
unsigned long long int T = 1024;

__global__ void square(unsigned long long int I[M], unsigned long long int O[N]){
	unsigned long long int x = blockIdx.x;
	unsigned long long int y = threadIdx.x;
	unsigned long long int size=I[0];	
	unsigned long long int T=1024;
	unsigned long long int B=1+(2*size-1)/T;
	unsigned long long int idx_start=(y+x*T);
	unsigned long long int idx_end=(1+y+x*T);
	long int C=B*T;

	/*if(2*size-1>C){
		int n=int((sqrtf(1+2*(2*size-1-C))-1)/2)+1;
		int rem=2*size-1-C-2*n*(n-1);
		int rem_l=rem/2;
		int rem_r=rem-rem_l;
		if(idx_start<n || idx_end>C-n){
			if(idx_start<C/2){
				int idx=idx_start;
				idx_start=(n*n-(n-idx)*(n-idx));
				idx_end=idx_start+(1+(n-idx-1)*2);
			}else{
				idx_start-=(C-n);
				int idx=idx_start;
				idx_start=(n*n+C-2*n+idx*idx);
				idx_end=idx_start+(1+idx*2);
			}
		}else{
			idx_start-=n;
			idx_start+=(n*n);
			idx_end-=n;
			idx_end+=(n*n);
		}	
		if(idx_start==0){
			idx_end+=rem_l;
		}else if(idx_end==2*size-1-(rem_l+rem_r)){
			idx_start+=rem_l;
			idx_end+=(rem_r+rem_l);
		}else{
			idx_start+=rem_l;
			idx_end+=rem_l;
		}
	}*/

	for(unsigned long long int idx=idx_start+1;idx<idx_end+1;idx++){
		if(idx<2*size){
			O[idx]=0;
			unsigned long long int i,j;
			if(idx<size+1){
				i=idx;
				j=1;
			}else{
				i=size;
				j=(idx%size)+1;
			}
			while(i!=0 && j!=size+1 && i>=j){
				if(i>j){
					O[idx]+=2*(I[i])*(I[j]);
				}
				else if(i==j){
					O[idx]+=(I[i])*(I[j]);
				}
				i--;
				j++;
			}
		}
	}

	//if(idx_end==2*size-1)
		// printf("index_end : %d\n", idx_end);
		// printf("%llu,%llu,%llu\n",B,I[0],T );
	// O[N-1]=1;
}

int main(){

	unsigned long long int *hostI=new unsigned long long int[M];
	unsigned long long int *hostO=new unsigned long long int[N];

	int total_count=1;
	int count=1;
	unsigned long long int size=M-1;
	unsigned long long int base=1024*1024;

	for(unsigned long long int i=0;i<size+1;i++){
		if(i==0){
			hostI[0]=size;
		}else if(i==size){
			hostI[i]=1;
		}else{
			hostI[i]=0;
		}
	}

	unsigned long long int *I;

	unsigned long long int *O;
	
	hipMalloc((void**)&I, sizeof(unsigned long long int) * M);

	hipMalloc((void**)&O, sizeof(unsigned long long int) * N);
	while(count<=total_count){
		hipError_t err=hipMemcpy(I,hostI,sizeof(unsigned long long int) * (size+1),hipMemcpyHostToDevice);
		cout<<err<<endl;

		err=hipMemcpy(O,hostO,sizeof(unsigned long long int) * (2*size),hipMemcpyHostToDevice);
		cout<<err<<endl;

		T=1024;
		B=1+(2*size-1)/T;

		dim3 blocks(B,1,1);
		dim3 threads(T,1,1);

		square<<<blocks,threads>>>((unsigned long long int(*))I, (unsigned long long int(*))O);	

		err=hipMemcpy(hostO,O,sizeof(unsigned long long int) * 2*size,hipMemcpyDeviceToHost);
		cout<<hipGetErrorString(err)<<endl;
		/*unsigned long long int c=0;
		int pos=1;
		int flag=0;
		while (c!=0 || pos<2*size){
			if(pos>=2*size)
				hostO[pos]=0;
			hostO[pos]=hostO[pos]+c;
			c=hostO[pos]/base;
			hostO[pos]=hostO[pos]%base;
			if(pos==1){
				if(hostO[pos]<2){
					hostO[pos]=base-2+hostO[pos];
					flag=1;
				}else{
					hostO[pos]-=2;
				}
			}else if(flag==1 && hostO[pos]==0){
				hostO[pos]=base-1;
			}else if(flag==1){
				hostO[pos]-=1;
				flag=0;
			}
			hostI[pos]=hostO[pos];
			// cout<<hostO[pos]<<",";
			hostO[pos]=0;
			if((pos>=2*size && (c!=0 || hostI[pos]!=0)) || pos<2*size)
				pos++;
		}
		if(hostI[pos-1]==0){
			pos--;
		}
		hostO[0]=pos-1;
		hostI[0]=hostO[0];
		size=hostI[0];
		hostO[0]=0;*/
		cout<<"size : "<<hostI[0]<<","<<hostO[2*size-1]<<endl;
	//for(int i=0;i<2*size;i++)
		//cout<<"working : "<<hostO[i]<<",";
		count++;
	}
	return 0;
}